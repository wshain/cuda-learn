
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
void cpu(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }
}
__global__ void gpu(int *a, int N)
{
    int threadi = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = threadi; i < N; i += stride)
    {
        a[i] *= 2;
    }
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA runtime error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}
int main()
{
    const int N = 1000;
    size_t size = N * sizeof(int);
    int *a;

    hipError_t err;
    err = hipMallocManaged(&a, size);
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    cpu(a, N);

    size_t threads = 256;
    size_t blocks = 1;
    gpu<<<blocks, -1>>>(a, N);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error gpu: %s\n", hipGetErrorString(err));
    }

    checkCuda(hipDeviceSynchronize());

    hipFree(a);
}