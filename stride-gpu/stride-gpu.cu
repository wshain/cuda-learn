// gridDim.x表示当前网格有几个block
// blockIdx.x表示当前第几个线程块，
// blockDim.x表示当前线程块内的线程数量
// threadIdx.x表示当前是第几个线程

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void cpu(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }
}
__global__ void gpu(int *a, int N)
{
    int threadi = blockIdx.x * blockDim.x + threadIdx.x; // 初始位置
    int stride = gridDim.x * blockDim.x;                 // 跨步大小
    for (int i = threadi; i < N; i += stride)
    {
        a[i] *= 2;
    }
}

bool check_cpu(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (a[i] != i)
            return false;
    }
    return true;
}
bool check_gpu(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}
int main()
{
    const int N = 1000;
    size_t size = N * sizeof(int);
    int *a;
    hipMallocManaged(&a, size); // 统一分配内存，可以被gpu使用也可以被cpu使用
    cpu(a, N);

    check_cpu(a, N) ? printf("cpu ok\n") : printf("cpu error\n");

    size_t threads = 256;
    size_t blocks = 1;
    gpu<<<blocks, threads>>>(a, N);
    hipDeviceSynchronize();

    check_gpu(a, N) ? printf("gpu ok") : printf("gpu error");

    hipFree(a);
}