#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
void cpu(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }
}
__global__ void gpu(int *a, int N)
{
    int threadi = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = threadi; i < N; i += stride)
    {
        a[i] *= 2;
    }
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA runtime error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}
int main()
{
    const int N = 1000;
    size_t size = N * sizeof(int);
    int *a;

    hipError_t err;
    err = hipMallocManaged(&a, size);
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    int id;
    hipGetDevice(&id);

    cpu(a, N); // 初始在cpu上

    hipMemPrefetchAsync(a, size, id); // 异步预取到gpu上，为后续的gpu()操作做准备，需要传入gpu的设备id

    size_t threads = 256;
    size_t blocks = 1;

    gpu<<<blocks, threads>>>(a, N);

    hipMemPrefetchAsync(a, size, hipCpuDeviceId); // 异步预取到cpu上，为的是后续的checkCuda函数，checkCuda函数是在cpu上进行，这时需传入cpu的设备id，直接传hipCpuDeviceId参数即可，不需要函数获取id
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error gpu: %s\n", hipGetErrorString(err));
    }

    checkCuda(hipDeviceSynchronize());

    hipFree(a);
}