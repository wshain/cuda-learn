
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64

__global__ void gpu(int *a, int *b, int *c_gpu)
{
    int r = blockDim.x * blockIdx.x + threadIdx.x;
    int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (r < N && c < N)
    {
        c_gpu[r * N + c] = a[r * N + c] + b[r * N + c];
    }
}

void cpu(int *a, int *b, int *c_cpu)
{
    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            c_cpu[r * N + c] = a[r * N + c] + b[r * N + c];
        }
}

bool check(int *a, int *b, int *c_cpu, int *c_gpu)
{
    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            if (c_cpu[r * N + c] != c_gpu[r * N + c])
                return false;
        }
    return true;
}
int main()
{
    int *a, *b, *c_cpu, *c_gpu, *c_gpu_cpu;
    size_t size = N * N * sizeof(int);

    hipHostMalloc(&a, size, hipHostMallocDefault);
    hipHostMalloc(&b, size, hipHostMallocDefault);
    hipHostMalloc(&c_cpu, size, hipHostMallocDefault);
    hipHostMalloc(&c_gpu_cpu, size, hipHostMallocDefault);
    hipMalloc(&c_gpu, size);

    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            a[r * N + c] = r;
            b[r * N + c] = c;
            c_cpu[r * N + c] = 0;
            // c_gpu[r * N + c] = 0;
            c_gpu_cpu[r * N + c] = 0;
        }

    cpu(a, b, c_cpu);

    dim3 threads(16, 16, 1);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y, 1);
    gpu<<<blocks, threads>>>(a, b, c_gpu);
    hipDeviceSynchronize();

    hipMemcpy(c_gpu_cpu, c_gpu, size, hipMemcpyDeviceToHost); // 把cudamalloc() 申请的数据从c_gpu 拷贝到c_gpu_cpu, 因为cudamalloc申请的内存只能gpu使用，cpu无法访问指针

    check(a, b, c_cpu, c_gpu_cpu) ? printf("ok") : printf("error");

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c_cpu);
    hipHostFree(c_gpu_cpu);
    hipFree(c_gpu);
}