//   .->MY
//  |
//  \/MX

#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define MX 2048
#define MY 2048

// native 版矩阵转置
__global__ void transpose(float *odata, float *idata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    if (x >= MX || y >= MY)
        return;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[x * width + (y + j)] = idata[(y + j) * width + x];
}

// 共享内存替换，带宽基本没有提升，因为要跨步拷贝
__global__ void shared_memory_transpose(float *odata, float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // 存储块冲突优化

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    if (x >= MX || y >= MY)
        return;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    __syncthreads();
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[x * width + y + j] = tile[threadIdx.y + j][threadIdx.x];
}

// 共享内存转置
__global__ void shared_memory_tr_transpose(float *odata, float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    if (x >= MX || y >= MY)
        return;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    __syncthreads();
    // 块位置交换,内部直接复制
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j]; // shared-mem转置
}

bool check(float *h_odata, float *res)
{
    for (int r = 0; r < MX; r++)
        for (int c = 0; c < MY; c++)
        {
            if (h_odata[r * MY + c] != res[r * MY + c])
                return false;
        }
    return true;
}
int main()
{
    size_t size = MX * MY * sizeof(float);
    float *h_idata, *h_odata, *d_idata, *d_odata, *res;
    hipHostMalloc(&h_idata, size, hipHostMallocDefault);
    hipHostMalloc(&h_odata, size, hipHostMallocDefault);
    hipHostMalloc(&res, size, hipHostMallocDefault);
    hipMalloc(&d_idata, size);
    hipMalloc(&d_odata, size);

    dim3 threads(TILE_DIM, BLOCK_ROWS, 1);
    // 上取整
    dim3 blocks((MX + TILE_DIM - 1) / TILE_DIM, (MY + TILE_DIM - 1) / TILE_DIM, 1);

    for (int r = 0; r < MX; r++)
        for (int c = 0; c < MY; c++)
        {
            h_idata[r * MY + c] = r * MY + c;
            res[r * MY + c] = c * MX + r;
        }

    hipEvent_t startEvent, stopEvent;
    float ms;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);

    shared_memory_tr_transpose<<<blocks, threads>>>(d_odata, d_idata);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);

    printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");
    printf("%25s", "native transpose");
    // 1GB = 1e3MB = 1e6KB = 1e9B学到了
    printf("%20.2f\n", 2 * MX * MY * sizeof(float) * 1e-9 / (ms / 1000));
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    check(h_odata, res) ? printf("ok\n") : printf("error\n");

    hipHostFree(h_idata);
    hipHostFree(h_odata);
    hipHostFree(res);
    hipFree(d_idata);
    hipFree(d_odata);
}