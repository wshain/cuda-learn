
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int id;
    hipGetDevice(&id);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, id);
    printf("device id: %d \n sms: %d \n capability major: %d \n capability minor: %d \n warp size: %d \n", id, props.multiProcessorCount, props.major, props.minor, props.warpSize);
}

// device id 设备id
// sms 处理器的个数
// （算力）架构：capabilities major整数范围 capabilities minor小数范围
// warp size ：warp大小
