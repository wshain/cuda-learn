
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000

__global__ void gpu(int *a, int *b, int *c_gpu)
{
    int r = blockDim.x * blockIdx.x + threadIdx.x;
    int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (r < N && c < N)
    {
        c_gpu[r * N + c] = a[r * N + c] + b[r * N + c];
    }
}

void cpu(int *a, int *b, int *c_cpu)
{
    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            c_cpu[r * N + c] = a[r * N + c] + b[r * N + c];
        }
}

bool check(int *c_cpu, int *c_gpu)
{
    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            if (c_cpu[r * N + c] != c_gpu[r * N + c])
                return false;
        }
    return true;
}
int main()
{
    int *a_cpu, *b_cpu, *a_gpu, *b_gpu, *c_cpu, *c_gpu, *c_gpu_cpu;
    size_t size = N * N * sizeof(int);

    hipHostMalloc(&a_cpu, size, hipHostMallocDefault);
    hipHostMalloc(&b_cpu, size, hipHostMallocDefault);
    hipMalloc(&a_gpu, size);
    hipMalloc(&b_gpu, size);
    hipHostMalloc(&c_cpu, size, hipHostMallocDefault);
    hipHostMalloc(&c_gpu_cpu, size, hipHostMallocDefault);
    hipMalloc(&c_gpu, size);

    for (int r = 0; r < N; r++)
        for (int c = 0; c < N; c++)
        {
            a_cpu[r * N + c] = r;
            b_cpu[r * N + c] = c;
            c_cpu[r * N + c] = 0;
            // c_gpu[r * N + c] = 0;
            c_gpu_cpu[r * N + c] = 0;
        }

    cpu(a_cpu, b_cpu, c_cpu);

    dim3 threads(16, 16, 1);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y, 1);

    hipStream_t s1, s2, s3, s4;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);
    hipStreamCreate(&s4);
    for (int i = 0; i < 20; i++)
    {
        hipMemcpyAsync(a_gpu, a_cpu, size, hipMemcpyDeviceToHost, s1);
        hipMemcpyAsync(b_gpu, b_cpu, size, hipMemcpyDeviceToHost, s2);

        gpu<<<blocks, threads, 0, s4>>>(a_gpu, b_gpu, c_gpu);

        hipMemcpyAsync(c_gpu_cpu, c_gpu, size, hipMemcpyDeviceToHost, s3); // 把cudamalloc() 申请的数据从c_gpu 拷贝到c_gpu_cpu, 因为cudamalloc申请的内存只能gpu使用，cpu无法访问指针
    }

    hipDeviceSynchronize();
    check(c_cpu, c_gpu_cpu) ? printf("ok") : printf("error");

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);
    hipStreamDestroy(s4);
    hipHostFree(a_cpu);
    hipHostFree(b_cpu);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipHostFree(c_cpu);
    hipHostFree(c_gpu_cpu);
    hipFree(c_gpu);
}